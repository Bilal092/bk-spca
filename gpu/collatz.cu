#include "hip/hip_runtime.h"
/*
 * Copyright Divakar Viswanath, 2009-2014
 */

/*     
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of version 2 of the GNU General Public License as 
 * published by the Free Software Foundation.

 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */

#ifndef NUMTHREADS
#define NUMTHREADS 384
#endif
#ifndef NUMBLOCKS
#define NUMBLOCKS 14
#endif
#define CLKSPD 1.147
#include <iostream>
#include <cassert>
#include <cstdlib>
using namespace std;
#include "dvhstmem.hh"
#include "hstTimer.hh"
#include "dvTimer.hh"

//list size must equal NUMTHREADS*32*NUMBLOCKS
//NUMTHREADS must be a multiple of 32
__global__ void 
__launch_bounds__(NUMTHREADS, 1536/NUMTHREADS)
collatzstride(int *list,  int niter){
  const int N = NUMTHREADS*32;
  int fst = N*blockIdx.x;
  int id = threadIdx.x;
  __shared__ int smemlist[N];
  for(int i=0; i < 32; i++)
    smemlist[NUMTHREADS*i+id] = list[fst+NUMTHREADS*i+id];
  
  for(int j=0; j < niter; j++)
    for(int i=0; i < 32; i++){//each thread works on 32 entries
      int x = smemlist[NUMTHREADS*i+id];
      if(x >= 0){
	x = 3*x+1;
	if(x > 1024*1024*256)
	  x=-1;
	else{
	  while(x%2==0)
	    x = x/2;
	}
	smemlist[NUMTHREADS*i+id] = x;
      }
    }
  for(int i=0; i < 32; i++)
    list[fst+NUMTHREADS*i+id] = smemlist[NUMTHREADS*i+id];
}

//list size must equal NUMTHREADS*32*NUMBLOCKS
//NUMTHREADS must be a multiple of 32
__global__ void 
__launch_bounds__(NUMTHREADS, 1536/NUMTHREADS)
collatzblock(int *list,  int niter){
  const int N = NUMTHREADS*32;
  int fst = N*blockIdx.x;
  int id = threadIdx.x;
  __shared__ int smemlist[N];
  for(int i=0; i < 32; i++)
    smemlist[NUMTHREADS*i+id] = list[fst+NUMTHREADS*i+id];
  
  for(int j=0; j < niter; j++)
    for(int i=0; i < 32; i++){//each thread works on 32 entries
      int x = smemlist[id*32+i];
      if(x >= 0){
	x = 3*x+1;
	if(x > 1024*1024*256)
	  x=-1;
	else{
	  while(x%2==0)
	    x = x/2;
	}
	smemlist[id*32+i] = x;
      }
    }
  for(int i=0; i < 32; i++)
    list[fst+NUMTHREADS*i+id] = smemlist[NUMTHREADS*i+id];
}

extern __shared__ char dynamicsharedmemory[];

//list size must equal NUMTHREADS*32*NUMBLOCKS
//NUMTHREADS must be a multiple of 32
__global__ void 
__launch_bounds__(NUMTHREADS, 1536/NUMTHREADS)
collatzdynamic(int *list,  int niter){
  const int N = NUMTHREADS*32;
  int fst = N*blockIdx.x;
  int id = threadIdx.x;
  int *smemlist;
  smemlist = (int *)(dynamicsharedmemory);
  for(int i=0; i < 32; i++)
    smemlist[NUMTHREADS*i+id] = list[fst+NUMTHREADS*i+id];
  
  for(int j=0; j < niter; j++)
    for(int i=0; i < 32; i++){//each thread works on 32 entries
      int x = smemlist[NUMTHREADS*i+id];
      if(x >= 0){
	x = 3*x+1;
	if(x > 1024*1024*512)
	  x=-1;
	else{
	  while(x%2==0)
	    x = x/2;
	}
	smemlist[NUMTHREADS*i+id] = x;
      }
    }
  for(int i=0; i < 32; i++)
    list[fst+NUMTHREADS*i+id] = smemlist[NUMTHREADS*i+id];
}

void runcollatz(int flag){
  dvhstmem<int> dhlist(NUMTHREADS*NUMBLOCKS*32);
  int *dlist = dhlist.device();
  int *hlist = dhlist.host();
  for(int i=0; i < NUMTHREADS*NUMBLOCKS*32; i++)
    hlist[i] = ((rand()*1.0)/RAND_MAX)*1000*1000;
  for(int i=0; i < 10; i++)
    cout<<"hlist[i] = "<<hlist[i]<<endl;
  cout<<endl;
  dhlist.host2device();
  int niter = 1000*1000;

  cout.width(30);
  cout<<"NUMTHREADS = "<<NUMTHREADS<<endl;
  cout.width(30);
  cout<<"NUMBLOCKS = "<<NUMBLOCKS<<endl;
  double telapsed;
  if(flag==0){
    hstTimer clk;
    clk.tic();
    collatzstride<<<NUMBLOCKS, NUMTHREADS>>>(dlist, niter);
    telapsed = clk.toc();
  }
  else if(flag==1){
    hstTimer clk;
    clk.tic();
    collatzblock<<<NUMBLOCKS, NUMTHREADS>>>(dlist, niter);
    telapsed = clk.toc();
  }
  else if(flag==2){
    hstTimer clk;
    clk.tic();
    collatzdynamic<<<NUMBLOCKS, NUMTHREADS, NUMTHREADS*32*4>>>(dlist, niter);
    telapsed = clk.toc();
  }
  cout.width(30);
  cout<<"telapsed = "<<telapsed<<" milliseconds"<<endl;
  cout.width(30);
  cout<<"iterations per cycle = "<<32.0*niter*NUMTHREADS*NUMBLOCKS/(telapsed*1e-3)/(CLKSPD*1e9)<<endl;
  cout<<endl;
  
  dhlist.device2host();
  for(int i=0; i < 10; i++)
    cout<<"hlist[i] = "<<hlist[i]<<endl;
  double avg = 0;
  for(int i=0; i < 32*NUMTHREADS*NUMBLOCKS; i++){
    if((hlist[i]!=1)&&(hlist[i]!=-1))
      cout<<"entry = "<<i<<" is neither 1 nor -1"<<endl;
    avg += hlist[i];
  }
  avg /= 32*NUMTHREADS*NUMBLOCKS;
  cout<<"avg = "<<avg<<endl;
}

int main(){
  runcollatz(0);
  runcollatz(1);
  runcollatz(2);
}
