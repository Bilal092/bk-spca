#include "hip/hip_runtime.h"
#ifndef NUMTHREADS
#define NUMTHREADS 384
#endif
#define CLKSPD 1.147
#include <iostream>
#include <cassert>
#include <cstdlib>
using namespace std;
#include "dvhstmem.hh"
#include "hstTimer.hh"
#include "dvTimer.hh"


__global__ void 
__launch_bounds__(NUMTHREADS, 1536/NUMTHREADS)
copystride(double *list, int n, double *copy){
  int tid = threadIdx.x + blockIdx.x*blockDim.x;
  int stride = blockDim.x*gridDim.x;
  for(int i=tid; i < n; i = i + stride)
    copy[i] = list[i]; 
}


void runcopystride(long int n, int NUMBLOCKS){
  double *dcopy, *hcopy;
  dvhstmem<double> dhcopy(n);
  dcopy = dhcopy.device();
  hcopy = dhcopy.host();
  double *hlist, *dlist;
  dvhstmem<double> dhlist(n);
  hlist = dhlist.host();
  dlist = dhlist.device();
  for(int i=0; i < n; i++)
    hlist[i] = rand()*1.0/RAND_MAX-0.5;
  dhlist.host2device();
  double telapsed = 1e100;
  for(int i=0; i < 10; i++){
    hstTimer hclk;
    hclk.tic();  
    copystride<<<NUMBLOCKS, NUMTHREADS>>>(dlist, n, dcopy);
#ifdef DEBUG
    printf("CUDA: %s\n", hipGetErrorString(hipGetLastError()));
#endif
    double tintvl=hclk.toc();
    cout.width(30);
    cout<<"tintvl = "<<tintvl<<endl;
    if(tintvl < telapsed)
      telapsed = tintvl;
  }
  
  dhcopy.device2host();
  double ans=0;
  for(int i=0; i < n; i++)
    ans += hcopy[i];
  cout.width(30);
  cout<<"List average = "<<ans/n<<endl;
  cout.width(30);
  cout<<"Time elapsed = "<<telapsed<<" milliseconds"<<endl;
  cout.width(30);
  cout<<"Bandwidth to memory = "<<2*n*8.0/(telapsed/1000)/(CLKSPD*1e9)<<" bytes/cycle"<<endl;
  cout.width(30);
  cout<<"Bandwidth to memory = "<<2*n*8.0/(telapsed/1000)/1e9<<" GB/s"<<endl;
}

int main(){
  int NUMBLOCKS = 56;
  int n = NUMTHREADS*NUMBLOCKS*4000;
  cout.width(30);
  cout<<"n = "<<n<<endl;
  cout.width(30);
  cout<<"NUMTHREADS = "<<NUMTHREADS<<endl;
  cout.width(30);
  cout<<"NUMBLOCKS = "<<NUMBLOCKS<<endl;
  runcopystride(n, NUMBLOCKS);
  cout<<endl<<endl;

  NUMBLOCKS = 56000;
  cout.width(30);
  cout<<"n = "<<n<<endl;
  cout.width(30);
  cout<<"NUMTHREADS = "<<NUMTHREADS<<endl;
  cout.width(30);
  cout<<"NUMBLOCKS = "<<NUMBLOCKS<<endl;
  runcopystride(n, NUMBLOCKS);
  cout<<endl<<endl;

}
