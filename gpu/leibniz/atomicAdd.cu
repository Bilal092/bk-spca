#include "hip/hip_runtime.h"
#include "atomicAdd.hh"
#include "../utils/const.hh"
//result and lock must be pointers to global memory 
//*result and *lock must be initialized to zero
__device__ void atomicAddDoubleDeadLock(double value, 
					volatile double *result, 
					volatile int *lock){
	int lockcopy=1;
	while(lockcopy==1)
		lockcopy = atomicExch((int *)lock, 1);
	*result += value;
	atomicExch((int *)lock, 0);
}


//result and lock must be pointers to global memory 
//*result and *lock must be initialized to zero
__device__ void atomicAddDouble(double value, 
				volatile double *result, volatile int *lock){
	for(int i=0; i < NWARP; i++){
		if(threadIdx.x%NWARP==i){
			int lockcopy=1;
			while(lockcopy==1)
				lockcopy = atomicExch((int *)lock, 1);
			*result += value;
			atomicExch((int *)lock, 0);
		}
	}
}

