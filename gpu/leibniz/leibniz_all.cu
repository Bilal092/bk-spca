#include "hip/hip_runtime.h"
/*
 * Copyright Divakar Viswanath, 2009-2014
 */

/*     
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of version 2 of the GNU General Public License as 
 * published by the Free Software Foundation.

 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */

#include "../utils/dhstmem.hh"
#include "../utils/hstTimer.hh"
#include "../utils/const.hh"
#include <iostream>
#include <cassert>
#include <cstdlib>


__global__ void 
__launch_bounds__(THinBLK, BLKinMP)
leibniz(long int n, double *result){
	int tid = threadIdx.x+blockIdx.x*blockDim.x;
	double ans=0;
	int step = blockDim.x*gridDim.x;
	for(long int i=tid; i < n; i+=step)
		ans = ans + 4.0/(2.0*i+1.0);
	if(tid%2==1)
		ans = -ans;
	result[tid] = ans;
}


//result and lock must be pointers to global memory 
//*result and *lock must be initialized to zero
__device__ void atomicAddDoubleDeadLock(double value, 
					volatile double *result, 
					volatile int *lock){
	int lockcopy=1;
	while(lockcopy==1)
		lockcopy = atomicExch((int *)lock, 1);
	*result += value;
	atomicExch((int *)lock, 0);
}


//result and lock must be pointers to global memory 
//*result and *lock must be initialized to zero
__device__ void atomicAddDouble(double value, 
				volatile double *result, volatile int *lock){
	for(int i=0; i < NWARP; i++){
		if(threadIdx.x%NWARP==i){
			int lockcopy=1;
			while(lockcopy==1)
				lockcopy = atomicExch((int *)lock, 1);
			*result += value;
			atomicExch((int *)lock, 0);
		}
	}
}



//result and lock must be initialized to zero.
__global__ void 
__launch_bounds__(THinBLK, BLKinMP)
leibniztotal(long int n, double* result, int* lock){
	int tid = threadIdx.x+blockIdx.x*blockDim.x;
	double ans=0;
	int step = blockDim.x*gridDim.x;
	for(long int i=tid; i < n; i+=step)
		ans = ans + 4.0/(2.0*i+1.0);
	if(tid%2==1)
		ans = -ans;
	atomicAddDouble(ans, result, lock);
}


__global__ void 
__launch_bounds__(THinBLK, BLKinMP)
leibnizfloat(int n, float *result){
	int tid = threadIdx.x+blockIdx.x*blockDim.x;
	float ans=0;
	int step = blockDim.x*gridDim.x;
	for(int i=tid; i < n; i+=step)
		ans = ans + 4.0f/(2.0f*i+1.0f);
	if(tid%2==1)
		ans = -ans;
	result[tid] = ans;
}

void run_leibniz(long int n){
	int NBLK = NMP*BLKinMP;
		
	double *dresult, *result;
	dhstmem<double> dhmem(THinBLK*NBLK);
	dresult = dhmem.device();
	result = dhmem.host();
	
	std::cout.width(40);
	std::cout<<std::endl<<"Leibniz partially"<<std::endl;
	
	hstTimer nvclk;
	nvclk.tic();  
	leibniz<<<NBLK, THinBLK>>>(n, dresult);
#ifdef DEBUG
	printf("CUDA: %s\n", hipGetErrorString(hipGetLastError()));
#endif
	double telapsed = nvclk.toc(); 
	dhmem.device2host();
	double ans=0;
	for(int i=0; i < NBLK*THinBLK; i++)
		ans += result[i];
	std::cout.width(30);
	std::cout<<"leibniz partial sum = "<<ans<<std::endl;
	std::cout.width(30);
	std::cout<<"THinBLK = "<<THinBLK<<std::endl;
	std::cout.width(30);
	std::cout<<"NBLK = "<<NBLK<<std::endl;
	std::cout.width(30);
	std::cout<<"Time elapsed = "<<telapsed<<" milliseconds"<<std::endl<<std::endl;
}

void run_leibniztotal(long int n){
	int NBLK = NMP*BLKinMP;
	
	double *dresult, *result;
	dhstmem<double> dhmem(1);
	dresult = dhmem.device();
	result = dhmem.host();
	*result = 0;
	dhmem.host2device();
	
	dhstmem<int> dhmemi(1);
	int *dlock = dhmemi.device();
	int *hlock = dhmemi.host();
	*hlock = 0;
	dhmemi.host2device();

	std::cout.width(40);
	std::cout<<std::endl<<"Leibniz in total"<<std::endl;

	hstTimer nvclk;
	nvclk.tic();  
	leibniztotal<<<NBLK,THinBLK>>>(n, dresult, dlock);
#ifdef DEBUG
	printf("CUDA: %s\n", hipGetErrorString(hipGetLastError()));
#endif
	std::cout.width(30);
	std::cout<<"Time elapsed = "<<nvclk.toc()<<" milliseconds"<<std::endl;
	dhmem.device2host();
	std::cout.width(30);
	std::cout<<"leibniz partial sum = "<<*result<<std::endl;
	std::cout.width(30);
	std::cout<<"THinBLK = "<<THinBLK<<std::endl;
	std::cout.width(30);
	std::cout<<"NBLK = "<<NBLK<<std::endl<<std::endl;
}

void run_leibnizfloat(int n){
	int NBLK = NMP*BLKinMP;
	
	float *dresult, *result;
	dhstmem<float> dhmem(THinBLK*NBLK);
	dresult = dhmem.device();
	result = dhmem.host();


	std::cout.width(40);
	std::cout<<"Leibniz in single precision"<<std::endl;

	hstTimer nvclk;
	nvclk.tic();  
	leibnizfloat<<<NBLK, THinBLK>>>(n, dresult);
#ifdef DEBUG
	printf("CUDA: %s\n", hipGetErrorString(hipGetLastError()));
#endif
	double telapsed = nvclk.toc(); 


	dhmem.device2host();
	float ans=0;
	for(int i=0; i < NBLK*THinBLK; i++)
		ans += result[i];

	std::cout.width(30);
	std::cout<<"leibniz partial sum = "<<ans<<std::endl;
	std::cout.width(30);
	std::cout<<"THinBLK = "<<THinBLK<<std::endl;
	std::cout.width(30);
	std::cout<<"NBLK = "<<NBLK<<std::endl;
	std::cout.width(30);
	std::cout<<"Time elapsed = "<<telapsed<<" milliseconds"
		 <<std::endl<<std::endl;
}

void run_all(long n){
	run_leibniz(n);
	run_leibniztotal(n);
	run_leibnizfloat(n);
}

int main(){
	long int n = 1000l*1000*1000*10;
	run_all(n);
}
