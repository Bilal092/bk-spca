#include "hip/hip_runtime.h"
#include "../utils/const.hh"
#include "atomicAdd.hh"
#include "leibniz.hh"


__global__ void 
__launch_bounds__(THinBLK, BLKinMP)
leibniz(long int n, double *result){
	int tid = threadIdx.x+blockIdx.x*blockDim.x;
	double ans=0;
	int step = blockDim.x*gridDim.x;
	for(long int i=tid; i < n; i+=step)
		ans = ans + 4.0/(2.0*i+1.0);
	if(tid%2==1)
		ans = -ans;
	result[tid] = ans;
}

//result and lock must be initialized to zero.
__global__ void 
__launch_bounds__(THinBLK, BLKinMP)
leibniztotal(long int n, double* result, int* lock){
	int tid = threadIdx.x+blockIdx.x*blockDim.x;
	double ans=0;
	int step = blockDim.x*gridDim.x;
	for(long int i=tid; i < n; i+=step)
		ans = ans + 4.0/(2.0*i+1.0);
	if(tid%2==1)
		ans = -ans;
	atomicAddDouble(ans, result, lock);
}

__global__ void 
__launch_bounds__(THinBLK, BLKinMP)
leibnizfloat(int n, float *result){
	int tid = threadIdx.x+blockIdx.x*blockDim.x;
	float ans=0;
	int step = blockDim.x*gridDim.x;
	for(int i=tid; i < n; i+=step)
		ans = ans + 4.0f/(2.0f*i+1.0f);
	if(tid%2==1)
		ans = -ans;
	result[tid] = ans;
}