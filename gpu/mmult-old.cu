#include "hip/hip_runtime.h"
/*
 * Copyright Divakar Viswanath, 2009-2014
 */

/*     
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of version 2 of the GNU General Public License as 
 * published by the Free Software Foundation.

 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */

#include <iostream>
#include <cassert>
#include <cstdlib>
using namespace std;
#include <dvhstmem.hh>
#include <hstTimer.hh>
const int NUMTHREADS=1024;
const int NUMBLOCKS=14*14*2;
const int N=8*8*14;
const int NW=8*14;
const double CLKSPD=1.147;

//C += A*B
//square matrices of dim 8*14*8=64*14
//blockDim.x==NUMTHREADS must be 1024
//gridDim.x==NUMBLOCKS must be 14*14*2
//number of warps  in grid = 32*14*14*2
__global__ void
__launch_bounds__(NUMTHREADS, 1)
mmult(double *A, double *B, double *C){
  int warpid = threadIdx.x/32+blockIdx.x*32;
  int tid32 = threadIdx.x%32;
  int ib = (warpid%NW);
  int jb = (warpid/NW);
  ib *= 8;
  jb *= 8;
  int i = ib + tid32%8;
  int j = jb + tid32/8;
  for(int k = 0; k < N; k++){
    C[i+j*N] += A[i+k*N]*B[k+j*N];
    C[i+(j+4)*N] += A[i+k*N]*B[k+(j+4)*N];
  }
}

//C += A*B
//square matrices of dim 8*14*8=64*14
//blockDim.x==NUMTHREADS must be 1024
//gridDim.x==NUMBLOCKS must be 14*14*2
//number of warps  in grid = 32*14*14*2
__global__ void
__launch_bounds__(NUMTHREADS, 1)
mmultsmem(double *A, double *B, double *C){
  int warpid = (threadIdx.x/32)+blockIdx.x*32;
  int tid32 = threadIdx.x%32;
  int warpidinblk = threadIdx.x/32;
  __shared__ double smem[3*64*32];
  double *smemfst = smem+warpidinblk*3*64;
  double *smemA = smemfst;
  double *smemB = smemfst + 64;
  double *smemC = smemfst + 128;
  int ib = warpid%NW;
  int jb = warpid/NW;
  ib *= 8;
  jb *= 8;
  int ii = tid32%8;
  int jj = tid32/8;
  int i = ib + ii;
  int j = jb + jj;
  smemC[ii+jj*8] = C[i+j*N];
  smemC[ii+(jj+4)*8] = C[i+(j+4)*N];
  for(int kb=0; kb < N; kb += 8){
    int kk = tid32/8;
    int k = kb + kk;
    smemA[ii+8*kk] = A[i+N*k];
    smemA[ii+8*(kk+4)] = A[i+N*(k+4)];
    kk = tid32%8;
    k = kb+kk;
    smemB[kk+8*jj] = B[k+j*N];
    smemB[kk+8*(jj+4)] = B[k+(j+4)*N];
    for(kk=0; kk < 8; kk++){
      smemC[ii+8*jj] += smemA[ii+8*kk]*smemB[kk+8*jj];
      smemC[ii+8*(jj+4)] += smemA[ii+8*kk]*smemB[kk+8*(jj+4)];
    }
  }
  C[i+j*N] = smemC[ii+jj*8];
  C[i+(j+4)*N] = smemC[ii+(jj+4)*8];
}

void testmmult(){
  dvhstmem<double> mA(N*N);
  dvhstmem<double> mB(N*N);
  dvhstmem<double> mC(N*N);
  double *A = mA.host();
  double *B = mB.host();
  double *C = mC.host();
  double *dA = mA.device();
  double *dB = mB.device();
  double *dC = mC.device();
  for(int  i=0; i < N; i++)
    for(int j=0; j < N; j++){
      A[i+j*N] = 1;
      B[i+j*N] = 1;
      C[i+j*N] = 0;
    }
  mA.host2device();
  mB.host2device();
  mC.host2device();
  mmultsmem<<<NUMBLOCKS, NUMTHREADS>>>(dA, dB, dC);
  printf("CUDA: %s\n", hipGetErrorString(hipGetLastError()));
  mmultsmem<<<NUMBLOCKS, NUMTHREADS>>>(dA, dB, dC);
  printf("CUDA: %s\n", hipGetErrorString(hipGetLastError()));
  mmultsmem<<<NUMBLOCKS, NUMTHREADS>>>(dA, dB, dC);
  printf("CUDA: %s\n", hipGetErrorString(hipGetLastError()));
  mC.device2host();
  cout<<"C[4] = "<<C[4]<<endl;
  cout<<"C[100004] = "<<C[100004]<<endl;
  for(int i=0; i < N*N; i++)
    if(C[i] != 2688)
      cout<<"error at i = "<<i<<" C[i] = "<<C[i]<<endl;
}

void runmmult(){
  dvhstmem<double> mA(N*N);
  dvhstmem<double> mB(N*N);
  dvhstmem<double> mC(N*N);
  double *A = mA.host();
  double *B = mB.host();
  double *C = mC.host();
  double *dA = mA.device();
  double *dB = mB.device();
  double *dC = mC.device();
  for(int  i=0; i < N; i++)
    for(int j=0; j < N; j++){
      A[i+j*N] = rand()*1.0/RAND_MAX-0.5;
      B[i+j*N] = rand()*1.0/RAND_MAX-0.5;
      C[i+j*N] = 0;
    }
  const int count = 100;
  hstTimer clk;
  clk.tic();
  for(int i=0; i < count; i++)
    mmult<<<NUMBLOCKS, NUMTHREADS>>>(dA, dB, dC);
  double telapsed = clk.toc();
  cout.width(40);
  cout<<"matrix multiplication in global memory"<<endl;
  cout.width(30);
  cout<<"count = "<<count<<endl;
  cout.width(30);
  cout<<"dim of square matrices N = "<<N<<endl;
  cout.width(30);
  cout<<"time elapsed = "<<telapsed<<" milliseconds"<<endl;
  cout.width(30);
  cout<<"giga flops per second = "<<2.0*count*N*N*N/(telapsed*1e-3)/1e9<<endl;
  cout.width(30);
  cout<<"flops per cycle = "<<2.0*count*N*N*N/(telapsed*1e-3)/(CLKSPD*1e9)<<endl;
  cout<<endl;
  
  for(int i=0; i < count; i++)
    mmultsmem<<<NUMBLOCKS, NUMTHREADS>>>(dA, dB, dC);
  telapsed = clk.toc();
  cout.width(40);
  cout<<"matrix multiplication using shared memory"<<endl;
  cout.width(30);
  cout<<"count = "<<count<<endl;
  cout.width(30);
  cout<<"dim of square matrices N = "<<N<<endl;
  cout.width(30);
  cout<<"time elapsed = "<<telapsed<<" milliseconds"<<endl;
  cout.width(30);
  cout<<"giga flops per second = "<<2.0*count*N*N*N/(telapsed*1e-3)/1e9<<endl;
  cout.width(30);
  cout<<"flops per cycle = "<<2.0*count*N*N*N/(telapsed*1e-3)/(CLKSPD*1e9)<<endl;
  cout<<endl;
  
}

int main(){
  //testmmult();
  runmmult();
}
