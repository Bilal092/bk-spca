/*
 * Copyright Divakar Viswanath, 2009-2014
 */

/*     
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of version 2 of the GNU General Public License as 
 * published by the Free Software Foundation.

 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */

#include <iostream>
using namespace std;

int main(){
	int count;
	hipGetDeviceCount(&count);
	cout.width(45);
	cout<<"Device Count: "<<count<<endl;
	for(int i=0; i < count; i++){
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		cout.width(45);
		cout<<"Device Name: "<<prop.name<<endl;
		cout.width(45);
		cout<<"Clock rate in GHz: "<<prop.clockRate*1.0/1000/1000<<endl;
		cout.width(45);
		cout<<"Major revision number: "<<prop.major<<endl;
		cout.width(45);
		cout<<"Minor revision number: "<<prop.minor<<endl;
		cout.width(45);
		cout<<"Global memory in GB: "
		    <<prop.totalGlobalMem*1.0/1000/1000/1000<<endl;
		cout.width(45);
		cout<<"Total L2 cache (in bytes): "
		    <<prop.l2CacheSize<<endl;
		cout.width(45);
		cout<<"Shared memory per SM (in bytes): "
		    <<prop.sharedMemPerBlock<<endl;
		cout.width(45);
		cout<<"Number of registers per SM: "<<prop.regsPerBlock<<endl;
		cout.width(45);
		cout<<"Max number of threads per SM: "
		    <<prop.maxThreadsPerMultiProcessor<<endl;
		cout.width(45);
		cout<<"Number of threads per warp: "<<prop.warpSize<<endl;
		cout.width(45);
		cout<<"Maximum number of threads per block: "
		    <<prop.maxThreadsPerBlock<<endl;
		cout.width(45);
		cout<<"Maximum of each dimension of block: "
		    <<prop.maxThreadsDim[0]<<" x "
		    <<prop.maxThreadsDim[1]<<" x "<<prop.maxThreadsDim[2]<<endl;
		cout.width(45);
		cout<<"Maximum of each dimension of grid: "
		    <<prop.maxGridSize[0]<<" x "
		    <<prop.maxGridSize[1]<<" x "<<prop.maxGridSize[1]<<endl;
		cout.width(45);
		cout<<"Overlap: ";
		cout<<((prop.deviceOverlap)?"enabled":"disabled")<<endl;
	}
}
