#include "hip/hip_runtime.h"
/*
 * Copyright Divakar Viswanath, 2009-2014
 */

/*     
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of version 2 of the GNU General Public License as 
 * published by the Free Software Foundation.

 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */

#ifndef NUMTHREADS
#define NUMTHREADS 384
#endif
#define CLKSPD 1.147
#include <iostream>
#include <cassert>
#include <cstdlib>
using namespace std;
#include "dvhstmem.hh"
#include "hstTimer.hh"
#include "dvTimer.hh"

__global__ void 
__launch_bounds__(NUMTHREADS, 1536/NUMTHREADS)
addblock(double *list, int n, double *result){
  int tid = threadIdx.x + blockIdx.x*blockDim.x;
  int blksize = n/(blockDim.x*gridDim.x);
  int fst = tid*blksize;
  int last = fst + blksize;
  double ans = 0;
  for(int i=fst; i < last; i++)
    ans += list[i];
  result[tid] = ans;
}

__global__ void 
__launch_bounds__(NUMTHREADS, 1536/NUMTHREADS)
addstride(double *list, int n, double *result){
  int tid = threadIdx.x + blockIdx.x*blockDim.x;
  int stride = blockDim.x*gridDim.x;
  double ans = 0;
  for(int i=tid; i < n; i = i + stride)
    ans += list[i]; 
  result[tid] = ans;
}

__global__ void 
__launch_bounds__(NUMTHREADS, 1536/NUMTHREADS)
addstride4(double *list, int n, double *result){
  int tid = threadIdx.x + blockIdx.x*blockDim.x;
  int stride = blockDim.x*gridDim.x;
  double ans0 = list[tid];
  double ans1 = list[tid+stride];
  double ans2 = list[tid+2*stride];
  double ans3 = list[tid+3*stride];
  result[tid] = ans0 + ans1 + ans2 + ans3;
}

void runaddstride(long int n, int NUMBLOCKS, int flag){
  double *dresult, *hresult;
  dvhstmem<double> dhresult(NUMTHREADS*NUMBLOCKS);
  dresult = dhresult.device();
  hresult = dhresult.host();
  double *hlist, *dlist;
  dvhstmem<double> dhlist(n);
  hlist = dhlist.host();
  dlist = dhlist.device();
  for(int i=0; i < n; i++)
    hlist[i] = rand()*1.0/RAND_MAX-0.5;//or try hlist[i]=i+1;
  dhlist.host2device();
  
  double telapsed = 1e100;
  for(int i = 0; i < 10; i++){//10 trials 
    double tintvl;
    hstTimer hclk;
    if(flag==-1){
      hclk.tic();  
      addblock<<<NUMBLOCKS, NUMTHREADS>>>(dlist, n, dresult);
#ifdef DEBUG
      printf("CUDA: %s\n", hipGetErrorString(hipGetLastError()));
#endif
      tintvl=hclk.toc();
    }
    else if(flag==0){
      hclk.tic(); 
      addstride<<<NUMBLOCKS, NUMTHREADS>>>(dlist, n, dresult);
#ifdef DEBUG
      printf("CUDA: %s\n", hipGetErrorString(hipGetLastError()));
#endif
      tintvl=hclk.toc();
    }
    else if(flag==1){
      hclk.tic(); 
      addstride4<<<NUMBLOCKS, NUMTHREADS>>>(dlist, n, dresult);
#ifdef DEBUG
      printf("CUDA: %s\n", hipGetErrorString(hipGetLastError()));
#endif
      tintvl=hclk.toc();
    }
    cout.width(30);
    cout<<"tintvl = "<<tintvl<<endl;
    if(tintvl < telapsed)
      telapsed = tintvl;
  }

  dhresult.device2host();
  double ans=0;
  for(int i=0; i < NUMBLOCKS*NUMTHREADS; i++)
    ans += hresult[i];
  cout.width(30);
  cout<<"List average = "<<ans/n<<endl;
  cout.width(30);
  cout<<"Time elapsed = "<<telapsed<<" milliseconds"<<endl;
  cout.width(30);
  cout<<"Bandwidth to memory = "<<(n+NUMTHREADS*NUMBLOCKS)*8.0/(telapsed/1000)/(CLKSPD*1e9)<<" bytes/cycle"<<endl;
  cout.width(30);
  cout<<"Bandwidth to memory = "<<(n+NUMTHREADS*NUMBLOCKS)*8.0/(telapsed/1000)/1e9<<" GB/s"<<endl;
}


int main(){
  int NUMBLOCKS = 56;
  int n = NUMTHREADS*NUMBLOCKS*4000;
  cout.width(30);
  cout<<"n = "<<n<<endl;
  cout.width(30);
  cout<<"NUMTHREADS = "<<NUMTHREADS<<endl;
  cout.width(30);
  cout<<"NUMBLOCKS = "<<NUMBLOCKS<<endl;
  cout.width(40);
  cout<<"add in blocks per thread"<<endl;
  runaddstride(n,NUMBLOCKS, -1);
  cout.width(40);
  cout<<"strided add"<<endl;
  runaddstride(n,NUMBLOCKS, 0);
  

  
  cout<<endl<<endl;
  NUMBLOCKS = 56000;
  n = NUMTHREADS*NUMBLOCKS*4;
  cout.width(30);
  cout<<"n = "<<n<<endl;
  cout.width(30);
  cout<<"NUMTHREADS = "<<NUMTHREADS<<endl;
  cout.width(30);
  cout<<"NUMBLOCKS = "<<NUMBLOCKS<<endl;
  cout.width(40);
  cout<<"add in blocks per thread"<<endl;
  runaddstride(n,NUMBLOCKS, -1);
  cout.width(40);
  cout<<"strided add"<<endl;
  runaddstride(n,NUMBLOCKS, 0);
  cout.width(40);
  cout<<"strided add 4"<<endl;
  runaddstride(n,NUMBLOCKS, 1);
}
