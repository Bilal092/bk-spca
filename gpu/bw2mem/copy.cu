#include "hip/hip_runtime.h"
#include "addcopy.hh"

__global__
__launch_bounds__(THinBLK, BLKinMP)
void copy(double *list, int n, double *copy){
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	int stride = blockDim.x*gridDim.x;
	for(int i=tid; i < n; i = i + stride)
		copy[i] = list[i]; 
}

