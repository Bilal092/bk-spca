#include "hip/hip_runtime.h"
#include "addcopy.hh"

__global__ 
__launch_bounds__(THinBLK, BLKinMP)
void addblock(double *list, int n, double *result){
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	int tcount = n/(blockDim.x*gridDim.x);
	int fst = tid*tcount;
	int last = fst + tcount;
	double ans = 0;
	for(int i=fst; i < last; i++)
		ans += list[i];
	result[tid] = ans;
}

__global__ 
__launch_bounds__(THinBLK, BLKinMP)
void addstride(double *list, int n, double *result){
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	int stride = blockDim.x*gridDim.x;
	double ans = 0;
	for(int i=tid; i < n; i = i + stride)
		ans += list[i]; 
	result[tid] = ans;
}



