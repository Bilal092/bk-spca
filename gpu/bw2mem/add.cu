#include "hip/hip_runtime.h"
/*
 * Copyright Divakar Viswanath, 2009-2014
 */

/*     
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of version 2 of the GNU General Public License as 
 * published by the Free Software Foundation.

 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */

#include "addcopy.hh"

__global__ 
__launch_bounds__(THinBLK, BLKinMP)
void addblock(double *list, int n, double *result){
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	int tcount = n/(blockDim.x*gridDim.x);
	int fst = tid*tcount;
	int last = fst + tcount;
	double ans = 0;
	for(int i=fst; i < last; i++)
		ans += list[i];
	result[tid] = ans;
}

__global__ 
__launch_bounds__(THinBLK, BLKinMP)
void addstride(double *list, int n, double *result){
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	int stride = blockDim.x*gridDim.x;
	double ans = 0;
	for(int i=tid; i < n; i = i + stride)
		ans += list[i]; 
	result[tid] = ans;
}


