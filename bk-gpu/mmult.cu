#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <cstdlib>
using namespace std;
#include <dvhstmem.hh>
#include <hstTimer.hh>
const int NUMTHREADS=1024;
const int NUMBLOCKS=14*14;
const int N=32*14;
const double CLKSPD=1.147;

__global__ void
__launch_bounds__(NUMTHREADS,1)
mmultgmem(double *A, double *B, double * C){
  int tid1024 = threadIdx.x;
  int ib = blockIdx.x%14;
  int jb = blockIdx.x/14;
  ib *= 32;
  jb *= 32;
  int i = ib + tid1024%32;
  int j = jb + tid1024/32;
  for(int k = 0; k < N; k++)
    C[i+j*N] += A[i+k*N]*B[k+j*N];
}

__global__ void
__launch_bounds__(NUMTHREADS,1)
mmultsmem(double *A, double *B, double * C){
  int tid1024 = threadIdx.x;
  __shared__ double smem[3*32*32];
  int ib = blockIdx.x%14;
  int jb = blockIdx.x/14;
  ib *= 32;
  jb *= 32;
  int ii = tid1024%32;
  int jj = tid1024/32;
  double *smemA = smem;
  double *smemB = smem+32*32;
  double *smemC = smem+2*32*32;
  smemC[ii+32*jj] = C[(ib+ii)+N*(jb+jj)];
  for(int kb = 0; kb < N; kb += 32){
    int kk = tid1024/32;
    smemA[ii+32*kk] = A[(ib+ii)+N*(kb+kk)]; 
    kk = tid1024%32;
    smemB[kk+32*jj] = B[(kb+kk)+N*(jb+jj)];
    __syncthreads();
    for(int kk=0; kk < 32; kk++)
      smemC[ii+32*jj] += smemA[ii+32*kk]*smemB[kk+32*jj];
  }
  C[(ib+ii)+N*(jb+jj)] = smemC[ii+32*jj];
}

void testmmult(){
  dvhstmem<double> mA(N*N);
  dvhstmem<double> mB(N*N);
  dvhstmem<double> mC(N*N);
  double *A = mA.host();
  double *B = mB.host();
  double *C = mC.host();
  double *dA = mA.device();
  double *dB = mB.device();
  double *dC = mC.device();
  for(int  i=0; i < N; i++)
    for(int j=0; j < N; j++){
      A[i+j*N] = 1;
      B[i+j*N] = 1;
      C[i+j*N] = 0;
    }
  mA.host2device();
  mB.host2device();
  mC.host2device();
  mmultgmem<<<NUMBLOCKS, NUMTHREADS>>>(dA, dB, dC);
  printf("CUDA: %s\n", hipGetErrorString(hipGetLastError()));
  mmultgmem<<<NUMBLOCKS, NUMTHREADS>>>(dA, dB, dC);
  printf("CUDA: %s\n", hipGetErrorString(hipGetLastError()));
  mmultgmem<<<NUMBLOCKS, NUMTHREADS>>>(dA, dB, dC);
  printf("CUDA: %s\n", hipGetErrorString(hipGetLastError()));
  mC.device2host();
  cout<<"C[4] = "<<C[4]<<endl;
  cout<<"C[100004] = "<<C[100004]<<endl;
  for(int i=0; i < N*N; i++)
    if(C[i] != 3*448){
      cout<<"error at i = "<<i<<" C[i] = "<<C[i]<<endl;
      break;
    }
}

void runmmult(char flag){
  dvhstmem<double> mA(N*N);
  dvhstmem<double> mB(N*N);
  dvhstmem<double> mC(N*N);
  double *A = mA.host();
  double *B = mB.host();
  double *C = mC.host();
  double *dA = mA.device();
  double *dB = mB.device();
  double *dC = mC.device();
  for(int  i=0; i < N; i++)
    for(int j=0; j < N; j++){
      A[i+j*N] = rand()*1.0/RAND_MAX-0.5;
      B[i+j*N] = rand()*1.0/RAND_MAX-0.5;
      C[i+j*N] = 0;
    }
  const int count = 1000;
  hstTimer clk;
  double telapsed;
  if(flag != 'g'){
    clk.tic();
    for(int i=0; i < count; i++)
      mmultsmem<<<NUMBLOCKS, NUMTHREADS>>>(dA, dB, dC);
    telapsed = clk.toc();
    cout.width(40);
    cout<<"matrix multiplication in shared memory"<<endl;
  }
  else{
    clk.tic();
    for(int i=0; i < count; i++)
      mmultgmem<<<NUMBLOCKS, NUMTHREADS>>>(dA, dB, dC);
    telapsed = clk.toc();
    cout.width(40);
    cout<<"matrix multiplication in global memory"<<endl;
  }
  cout.width(30);
  cout<<"count = "<<count<<endl;
  cout.width(30);
  cout<<"dim of square matrices N = "<<N<<endl;
  cout.width(30);
  cout<<"time elapsed = "<<telapsed<<" milliseconds"<<endl;
  cout.width(30);
  cout<<"giga flops per second = "<<2.0*count*N*N*N/(telapsed*1e-3)/1e9<<endl;
  cout.width(30);
  cout<<"flops per cycle = "<<2.0*count*N*N*N/(telapsed*1e-3)/(CLKSPD*1e9)<<endl;
  cout<<endl;
}

int main(){
  testmmult();
  runmmult('s');
  runmmult('g');
}
