#include "hip/hip_runtime.h"
#ifndef NUMTHREADS
#define NUMTHREADS 1024
#endif
#ifndef NUMBLOCKS
#define NUMBLOCKS 16
#endif
#define CLKSPD 1.301
#include <iostream>
#include <cassert>
#include <cstdlib>
using namespace std;
#include "dvhstmem.hh"
#include "hstTimer.hh"
#include "dvTimer.hh"


__global__ void 
__launch_bounds__(NUMTHREADS, NUMBLOCKS/16)
leibniz(long int n, double *result){
  int tid = threadIdx.x+blockIdx.x*blockDim.x;
  double ans=0;
  int step = blockDim.x*gridDim.x;
  for(long int i=tid; i < n; i+=step)
    ans = ans + 4.0/(2.0*i+1.0);
  if(tid%2==1)
    ans = -ans;
  result[tid] = ans;
}


//result and lock must be pointers to global memory 
//*result and *lock must be initialized to zero
__device__ void atomicAddDoubleDeadLock(double value, volatile double *result, volatile int *lock){
  int lockcopy=1;
  while(lockcopy==1)
    lockcopy = atomicExch((int *)lock, 1);
  *result += value;
  atomicExch((int *)lock, 0);
}


//result and lock must be pointers to global memory 
//*result and *lock must be initialized to zero
__device__ void atomicAddDouble(double value, volatile double *result, volatile int *lock){
  for(int i=0; i < 32; i++){
    if(threadIdx.x%32==i){
      int lockcopy=1;
      while(lockcopy==1)
	lockcopy = atomicExch((int *)lock, 1);
      *result += value;
      atomicExch((int *)lock, 0);
    }
  }
}

//result and lock must be initialized to zero.
__global__ void 
__launch_bounds__(NUMTHREADS, NUMBLOCKS/16)
leibniztotal(long int n, double* result, int* lock){
  int tid = threadIdx.x+blockIdx.x*blockDim.x;
  double ans=0;
  int step = blockDim.x*gridDim.x;
  for(long int i=tid; i < n; i+=step)
    ans = ans + 4.0/(2.0*i+1.0);
  if(tid%2==1)
    ans = -ans;
  atomicAddDouble(ans, result, lock);
}

__global__ void 
__launch_bounds__(NUMTHREADS, NUMBLOCKS/16)
leibnizfloat(int n, float *result){
  int tid = threadIdx.x+blockIdx.x*blockDim.x;
  float ans=0;
  int step = blockDim.x*gridDim.x;
  for(int i=tid; i < n; i+=step)
    ans = ans + 4.0f/(2.0f*i+1.0f);
  if(tid%2==1)
    ans = -ans;
  result[tid] = ans;
}

void runleibniz(long int n){
  double *dresult, *result;
  dvhstmem<double> dhmem(NUMTHREADS*NUMBLOCKS);
  dresult = dhmem.device();
  result = dhmem.host();


  hstTimer nvclk;
  nvclk.tic();  
  leibniz<<<NUMBLOCKS, NUMTHREADS>>>(n, dresult);
#ifdef DEBUG
    printf("CUDA: %s\n", hipGetErrorString(hipGetLastError()));
#endif
  double telapsed = nvclk.toc(); 
  dhmem.device2host();
  double ans=0;
  for(int i=0; i < NUMBLOCKS*NUMTHREADS; i++)
    ans += result[i];
  cout.width(30);
  cout<<"leibniz partial sum = "<<ans<<endl;
  cout.width(30);
  cout<<"NUMTHREADS = "<<NUMTHREADS<<endl;
  cout.width(30);
  cout<<"NUMBLOCKS = "<<NUMBLOCKS<<endl;
  cout.width(30);
  cout<<"Time elapsed = "<<telapsed<<" milliseconds"<<endl;
  cout.width(30);
  cout<<"Cycles per term = "<<telapsed/n*(CLKSPD*1e6)<<endl;
  cout.width(30);
  cout<<"Warp cycles per term = "<<
    telapsed/n*(CLKSPD*1e6)*NUMTHREADS*NUMBLOCKS/32<<endl;
}

void runleibniztotal(long int n){
  double *dresult, *result;
  dvhstmem<double> dhmem(1);
  dresult = dhmem.device();
  result = dhmem.host();
  *result = 0;
  dhmem.host2device();

  dvhstmem<int> dhmemi(1);
  int *dlock = dhmemi.device();
  int *hlock = dhmemi.host();
  *hlock = 0;
  dhmemi.host2device();

  cout.width(40);
  cout<<endl<<"Leibniz in blocks"<<endl;
  hstTimer nvclk;
  nvclk.tic();  
  leibniztotal<<<NUMBLOCKS,NUMTHREADS>>>(n, dresult, dlock);
#ifdef DEBUG
  printf("CUDA: %s\n", hipGetErrorString(hipGetLastError()));
#endif
  cout.width(30);
  cout<<"Time elapsed = "<<nvclk.toc()<<" milliseconds"<<endl;
  dhmem.device2host();
  cout.width(30);
  cout<<"leibniz partial sum = "<<*result<<endl;
}

void runleibnizfloat(int n){
  float *dresult, *result;
  dvhstmem<float> dhmem(NUMTHREADS*NUMBLOCKS);
  dresult = dhmem.device();
  result = dhmem.host();
  cout<<endl;
  cout.width(40);
  cout<<"Leibniz in single precision"<<endl;
  hstTimer nvclk;
  nvclk.tic();  
  leibnizfloat<<<NUMBLOCKS, NUMTHREADS>>>(n, dresult);
#ifdef DEBUG
    printf("CUDA: %s\n", hipGetErrorString(hipGetLastError()));
#endif
  double telapsed = nvclk.toc(); 
  dhmem.device2host();
  float ans=0;
  for(int i=0; i < NUMBLOCKS*NUMTHREADS; i++)
    ans += result[i];
  cout.width(30);
  cout<<"leibniz partial sum = "<<ans<<endl;
  cout.width(30);
  cout<<"NUMTHREADS = "<<NUMTHREADS<<endl;
  cout.width(30);
  cout<<"NUMBLOCKS = "<<NUMBLOCKS<<endl;
  cout.width(30);
  cout<<"Time elapsed = "<<telapsed<<" milliseconds"<<endl;
  cout.width(30);
  cout<<"Cycles per term = "<<telapsed/n*(CLKSPD*1e6)<<endl;
  cout.width(30);
  cout<<"Warp cycles per term = "<<
    telapsed/n*(CLKSPD*1e6)*NUMTHREADS*NUMBLOCKS/32<<endl;
}

int main(){
  long int n = 1000*1000*1000;
  n = n*10;
  //runleibniz(n);
  runleibniztotal(n);
  //int m = 1024*1024*512*3;
  //runleibnizfloat(m);
}
